#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include "cudarand2.hpp"

*cudarand( int *final,long size,long  max_value  )
{
    int seed =123;
  // setup execution grid to get max performance
  threadsX = 192;
  blocksX  = 32;

  const int nThreads = threadsX*blocksX;

  uint2* seeds = new uint2[ nThreads ];

  cutilSafeCall( hipMalloc( (void**) &state, sizeof(uint2)*nThreads ) );

  // calculate strided iteration constants
  unsigned long long A, C;
  A = 1LL; C = 0LL;
  for (unsigned int i = 0; i < nThreads; ++i) {
    C += A*c;
    A *= a;
  }
  A0 = A & 0xFFFFFFLL;
  A1 = (A >> 24) & 0xFFFFFFLL;
  C0 = C & 0xFFFFFFLL;
  C1 = (C >> 24) & 0xFFFFFFLL;

  // prepare first nThreads random numbers from seed
  unsigned long long x = (((unsigned long long)seed) << 16) | 0x330E;
  for (unsigned int i = 0; i < nThreads; ++i) {
    x = a*x + c;
    seeds[i].x = x & 0xFFFFFFLL;
    seeds[i].y = (x >> 24) & 0xFFFFFFLL;
  }

  cutilSafeCall(hipMemcpy(state, seeds, sizeof(uint2)*nThreads, hipMemcpyHostToDevice));

  delete[] seeds;
  generate(size, max_value);
  get(final,size);
  cleanup();
  return final;
}

void
//RNG_rand48::cleanup() {
  cleanup(){
  cutilSafeCall(hipFree((void*) state));
  cutilSafeCall(hipFree((void*) res));
}

void
//RNG_rand48::generate(int n)
generate(long n, long max_value)
{
  const int nThreads = threadsX*blocksX;

  int num_blocks = (n + nThreads-1)/nThreads;
	
  if (res == 0) {
    cutilSafeCall(hipMalloc( (void**) &res, sizeof(int)*nThreads*num_blocks));
  }
  
  dim3 grid( blocksX, 1, 1);
  dim3 threads( threadsX, 1, 1);

  uint2 A, C;
  A.x = A0; A.y = A1;
  C.x = C0; C.y = C1;

  // call GPU kernel
  RNG_rand48_get_int<<< grid, threads >>>((uint2 *)state, (int *)res, num_blocks, A, C, max_value);
}

void
//RNG_rand48::get(int *r, int n)
get(int *r, int n)
 {
  cutilSafeCall(hipMemcpy( r, res, sizeof(int)*n, hipMemcpyDeviceToHost ) );
}
