#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#define BLOCK_SIZE 16
#include <stdio.h>

#if defined(USE_POW)
#define r23 pow(0.5, 23.0)
#define r46 (r23*r23)
#define t23 pow(2.0, 23.0)
#define t46 (t23*t23)
#else
#define r23 (0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5*0.5)
#define r46 (r23*r23)
#define t23 (2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0*2.0)
#define t46 (t23*t23)
#endif
#define	A_VAR	1220703125.0

int *cudarand(int h_A[], int N);
 __device__ double	randlcK(double *X, double *A);
__device__ double   find_my_seedK( long kn,       /* my processor rank, 0<=kn<=num procs */
                       long np,       /* np = num procs                      */
                       long nn);       /* total num of ran numbers, all procs */

 __device__ double	randlcK(double *X, double *A)
{
       int      KS=0;
       double	R23, R46, T23, T46;
      double		T1, T2, T3, T4;
      double		A1;
      double		A2;
      double		X1;
      double		X2;
      double		Z;
      int     		i, j;

      if (KS == 0) 
      {
        R23 = 1.0;
        R46 = 1.0;
        T23 = 1.0;
        T46 = 1.0;
    
        for (i=1; i<=23; i++)
        {
          R23 = 0.50 * R23;
          T23 = 2.0 * T23;
        }
        for (i=1; i<=46; i++)
        {
          R46 = 0.50 * R46;
          T46 = 2.0 * T46;
        }
        KS = 1;
      }

/*  Break A into two parts such that A = 2^23 * A1 + A2 and set X = N.  */

      T1 = R23 * *A;
      j  = T1;
      A1 = j;
      A2 = *A - T23 * A1;

/*  Break X into two parts such that X = 2^23 * X1 + X2, compute
    Z = A1 * X2 + A2 * X1  (mod 2^23), and then
    X = 2^23 * Z + A2 * X2  (mod 2^46).                            */

      T1 = R23 * *X;
      j  = T1;
      X1 = j;
      X2 = *X - T23 * X1;
      T1 = A1 * X2 + A2 * X1;
      
      j  = R23 * T1;
      T2 = j;
      Z = T1 - T23 * T2;
      T3 = T23 * Z + A2 * X2;
      j  = R46 * T3;
      T4 = j;
      *X = T3 - T46 * T4;
      return(R46 * *X);
}
__device__ double   find_my_seedK( long kn,       /* my processor rank, 0<=kn<=num procs */
                       long np,       /* np = num procs                      */
                       long nn)       /* total num of ran numbers, all procs */
                      // double s,      /* Ran num seed, for ex.: 314159265.00 */
                      // double a )     /* Ran num gen mult, try 1220703125.00 */
{
    double s=314159265.00;
    double a=122073125.00;
      double t1,t2;
      long   mq,nq,kk,ik;
      if ( kn == 0 ) return s;
      mq = (nn/4 + np - 1) / np;
      nq = mq * 4 * kn;               /* number of rans to be skipped */
      t1 = s;
      t2 = a;
      kk = nq;
      while ( kk > 1 ) {
      	 ik = kk / 2;
         if( 2 * ik ==  kk ) {
            (void)randlcK( &t2, &t2 );
	    kk = ik;
	 }
	 else {
            (void)randlcK( &t1, &t2 );
	    kk = kk - 1;
	 }
      }
      (void)randlcK( &t1, &t2 );
      return( t1 );
}



__global__ void cudarandK( int* y,long n,long max_key) {

    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    //Total number of threads in execution grid
    const int THREAD_N = blockDim.x * gridDim.x;

    //No matter how small is execution grid or how large OptN is,
    //exactly OptN indices will be processed with perfect memory coalescing
    for(int opt = tid; opt < n; opt += THREAD_N){
	double x;
    double a=1220703125.00;                   /* Random number gen mult */
    double seed;
        seed= find_my_seedK(opt,THREAD_N,n);
        //printf("seed=%f\n",seed);
	    x = randlcK(&seed, &a);
	    x += randlcK(&seed, &a);
    	    x += randlcK(&seed, &a);
	    x += randlcK(&seed, &a);  
            y[opt] = max_key*x/4;
        }
}

int *cudarand(int h_A[], long N, long max_key) {

    int* d_A;
size_t size = N*sizeof(int);
   cutilSafeCall(
         hipMalloc((void**)&d_A, size) );
    // Copy vectors from host memory to device memory


    cudarandK<<<480, 256>>>(d_A, N,max_key);
     cutilCheckMsg("kernel launch failure");
   cutilSafeCall(
     hipDeviceSynchronize() );

    // Copy result from device memory to host memory
    // h1_C contains the result in host memory
   cutilSafeCall(
     hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost) );

        cutilSafeCall(hipFree(d_A));
        cutilSafeCall(hipDeviceReset());
    return h_A;
}



