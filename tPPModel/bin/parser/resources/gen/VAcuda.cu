#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cutil_inline.h>


void extern timerstart(char* name);
void extern timerend();

// Device code
__device__ void vectorAdd_main(int* A,int* B,int* C, int opt)
{
/* Sample Vector add code 
 C[opt]=A[opt]+ B[opt];
*/


}

//This kernel distributes the work irrespective of the size
__global__ void vectorAdd_kernel(int* A,int* B,int* C,int TOTAL_SIZE)
{
    const int      tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int THREAD_N = blockDim.x * gridDim.x;

    for(int opt = tid; opt < TOTAL_SIZE; opt += THREAD_N){
          vectorAdd_main(A,B,C,opt);    
	}
}

// Host code
int* abstract_vectorAdd(int* A,int* B,int* C,int TOTAL_SIZE)
{
	    timerstart("CUDA");
int* d_A ; 
int* d_B ; 
int* d_C ; 

    // Allocate vectors in device memory
    cutilSafeCall( hipMalloc((void**)&d_A, sizeof(int)*1000)) ; 
    cutilSafeCall( hipMalloc((void**)&d_B, sizeof(int)*1000)) ; 
    cutilSafeCall( hipMalloc((void**)&d_C, sizeof(int)*1000)) ; 

    // Copy variables from host memory to device memory

    cutilSafeCall( hipMemcpy(d_A,A, sizeof(int)*1000,hipMemcpyHostToDevice) ); 
    cutilSafeCall( hipMemcpy(d_B,B, sizeof(int)*1000,hipMemcpyHostToDevice) ); 
    cutilSafeCall( hipMemcpy(d_C,C, sizeof(int)*1000,hipMemcpyHostToDevice) ); 

    // Kernel call with 480*256 threads



    vectorAdd_kernel<<<480, 256>>>(d_A,d_B,d_C,TOTAL_SIZE);


    cutilCheckMsg("kernel launch failure\n");
    cutilSafeCall( hipDeviceSynchronize() );

    // Copy variables from device memory to host memory

    cutilSafeCall( hipMemcpy(C,d_C, sizeof(int)*1000,hipMemcpyDeviceToHost) ); 

      if (d_A) cutilSafeCall( hipFree(d_A)) ; 
      if (d_B) cutilSafeCall( hipFree(d_B)) ; 
      if (d_C) cutilSafeCall( hipFree(d_C)) ; 
      timerend();

    return C;

}